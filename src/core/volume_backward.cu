#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <cstring>
#include "dvren.h"

using namespace dvren;

namespace
{
    struct HostCtx
    {
        int width;
        int height;
        int n_steps;
        float dt;
        float sigma_scale;
        float stop_thresh;
        FieldType type;
        void* device_ctx;
        const float* d_origins;
        const float* d_dirs;
        const float* d_tmin;
    };

    struct GridDenseDev
    {
        int nx, ny, nz;
        float bmin[3];
        float bmax[3];
        float* sigma;
        float* rgb;
        float* g_sigma;
        float* g_rgb;
    };

    __device__ inline float clampf(float x, float a, float b) { return x < a ? a : (x > b ? b : x); }

    __device__ inline void grid_weights(const GridDenseDev* g, const float* p, int idxs[8], float w[8])
    {
        float ux = (p[0] - g->bmin[0]) / (g->bmax[0] - g->bmin[0]);
        float uy = (p[1] - g->bmin[1]) / (g->bmax[1] - g->bmin[1]);
        float uz = (p[2] - g->bmin[2]) / (g->bmax[2] - g->bmin[2]);
        ux = clampf(ux, 0.f, 1.f);
        uy = clampf(uy, 0.f, 1.f);
        uz = clampf(uz, 0.f, 1.f);
        float x = ux * (g->nx - 1), y = uy * (g->ny - 1), z = uz * (g->nz - 1);
        int x0 = (int)floorf(x), y0 = (int)floorf(y), z0 = (int)floorf(z);
        int x1 = x0 + 1 < g->nx ? x0 + 1 : x0, y1 = y0 + 1 < g->ny ? y0 + 1 : y0, z1 = z0 + 1 < g->nz ? z0 + 1 : z0;
        float tx = x - x0, ty = y - y0, tz = z - z0;
        auto idx = [&](int xi, int yi, int zi) { return (zi * g->ny + yi) * g->nx + xi; };
        int i000 = idx(x0, y0, z0), i100 = idx(x1, y0, z0), i010 = idx(x0, y1, z0), i110 = idx(x1, y1, z0);
        int i001 = idx(x0, y0, z1), i101 = idx(x1, y0, z1), i011 = idx(x0, y1, z1), i111 = idx(x1, y1, z1);
        float wx0 = 1.f - tx, wx1 = tx, wy0 = 1.f - ty, wy1 = ty, wz0 = 1.f - tz, wz1 = tz;
        idxs[0] = i000;
        idxs[1] = i100;
        idxs[2] = i010;
        idxs[3] = i110;
        idxs[4] = i001;
        idxs[5] = i101;
        idxs[6] = i011;
        idxs[7] = i111;
        w[0] = wx0 * wy0 * wz0;
        w[1] = wx1 * wy0 * wz0;
        w[2] = wx0 * wy1 * wz0;
        w[3] = wx1 * wy1 * wz0;
        w[4] = wx0 * wy0 * wz1;
        w[5] = wx1 * wy0 * wz1;
        w[6] = wx0 * wy1 * wz1;
        w[7] = wx1 * wy1 * wz1;
    }

    __device__ inline void sample_sigma_rgb(const GridDenseDev* g, const float* p, float& sig, float col[3])
    {
        int idxs[8];
        float w[8];
        grid_weights(g, p, idxs, w);
        float s = 0.f, r = 0.f, gg = 0.f, b = 0.f;
        for (int k = 0; k < 8; ++k)
        {
            int i = idxs[k];
            float wk = w[k];
            s += wk * g->sigma[i];
            r += wk * g->rgb[i * 3 + 0];
            gg += wk * g->rgb[i * 3 + 1];
            b += wk * g->rgb[i * 3 + 2];
        }
        sig = s;
        col[0] = r;
        col[1] = gg;
        col[2] = b;
    }

    __global__ void k_backward_grid_dense(
        const float* origins, const float* dirs, const float* tmin,
        int W, int H, int n_steps, float dt, float ksig, float stop_thresh,
        const float* dL_dimg, GridDenseDev* g)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= W || y >= H) return;
        int pix = y * W + x;
        const float* o = origins + pix * 3;
        const float* d = dirs + pix * 3;
        float t0 = tmin[pix];
        float3 gC = make_float3(dL_dimg[pix * 3 + 0], dL_dimg[pix * 3 + 1], dL_dimg[pix * 3 + 2]);
        const int S = 1024;
        int steps = n_steps;
        if (steps > S) steps = S;
        float a_buf[S];
        float T_buf[S];
        int used = 0;
        float T = 1.f;
        for (int i = 0; i < steps; ++i)
        {
            float t = t0 + dt * i;
            float p[3] = {o[0] + t * d[0], o[1] + t * d[1], o[2] + t * d[2]};
            float sig;
            float col[3];
            sample_sigma_rgb(g, p, sig, col);
            float a = 1.f - expf(-ksig * sig * dt);
            a_buf[i] = a;
            T_buf[i] = T;
            T *= 1.f - a;
            used = i + 1;
            if (T < stop_thresh) break;
        }
        float dT_next = 0.f;
        for (int i = used - 1; i >= 0; --i)
        {
            float T_i = T_buf[i];
            float a = a_buf[i];
            float t = t0 + dt * i;
            float p[3] = {o[0] + t * d[0], o[1] + t * d[1], o[2] + t * d[2]};
            float sig;
            float col[3];
            sample_sigma_rgb(g, p, sig, col);
            float w_i = T_i * a;
            float3 dL_dc = make_float3(gC.x * w_i, gC.y * w_i, gC.z * w_i);
            float dL_dw = gC.x * col[0] + gC.y * col[1] + gC.z * col[2];
            float dL_dTi = dL_dw * a + dT_next * (1.f - a);
            float dL_dai = dL_dw * T_i + dT_next * (-T_i);
            float dai_dsigma = ksig * dt * (1.f - a);
            float dL_dsigma = dL_dai * dai_dsigma;
            int idxs[8];
            float ww[8];
            grid_weights(g, p, idxs, ww);
            for (int k = 0; k < 8; ++k)
            {
                int vi = idxs[k];
                float wk = ww[k];
                atomicAdd(&g->g_sigma[vi], dL_dsigma * wk);
                atomicAdd(&g->g_rgb[vi * 3 + 0], dL_dc.x * wk);
                atomicAdd(&g->g_rgb[vi * 3 + 1], dL_dc.y * wk);
                atomicAdd(&g->g_rgb[vi * 3 + 2], dL_dc.z * wk);
            }
            dT_next = dL_dTi;
        }
    }
}

bool dvren::volume_backward(void* saved_ctx, size_t saved_ctx_bytes, const float* dL_dimage, int width, int height, const FieldProvider& field)
{
    if (field.type != Field_Grid_Dense) return false;
    HostCtx h{};
    hipMemcpy(&h, saved_ctx, sizeof(HostCtx), hipMemcpyDeviceToHost);
    if (h.width != width || h.height != height) return false;
    auto g_dev = reinterpret_cast<GridDenseDev*>(field.device_ctx);
    GridDenseDev g_host{};
    hipMemcpy(&g_host, g_dev, sizeof(GridDenseDev), hipMemcpyDeviceToHost);
    if (!g_host.g_sigma || !g_host.g_rgb) return false;
    float* d_grad = nullptr;
    size_t n = static_cast<size_t>(width) * height * 3;
    hipMalloc(&d_grad, sizeof(float) * n);
    hipMemcpy(d_grad, dL_dimage, sizeof(float) * n, hipMemcpyHostToDevice);
    dim3 bs(16, 16, 1);
    dim3 gs((width + bs.x - 1) / bs.x, (height + bs.y - 1) / bs.y, 1);
    k_backward_grid_dense<<<gs,bs>>>(
        h.d_origins, h.d_dirs, h.d_tmin,
        width, height, h.n_steps, h.dt, h.sigma_scale, h.stop_thresh,
        d_grad, g_dev);
    hipDeviceSynchronize();
    hipFree(d_grad);
    return true;
}
