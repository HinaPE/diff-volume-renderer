#include <hip/hip_runtime.h>
#include <vector>
#include <cstring>
#include "dvren.h"

using namespace dvren;

namespace
{
    struct GridDenseDev
    {
        int nx, ny, nz;
        float bmin[3];
        float bmax[3];
        float* sigma;
        float* rgb;
        float* g_sigma;
        float* g_rgb;
    };
}

static size_t voxels_count(const GridDenseDev& g)
{
    return (size_t)g.nx * g.ny * g.nz;
}

bool dvren::field_grid_dense_create(const GridDenseDesc& desc, FieldProvider& out)
{
    GridDenseDev h{};
    h.nx = desc.nx;
    h.ny = desc.ny;
    h.nz = desc.nz;
    h.bmin[0] = desc.bbox_min[0];
    h.bmin[1] = desc.bbox_min[1];
    h.bmin[2] = desc.bbox_min[2];
    h.bmax[0] = desc.bbox_max[0];
    h.bmax[1] = desc.bbox_max[1];
    h.bmax[2] = desc.bbox_max[2];
    size_t vox = (size_t)desc.nx * desc.ny * desc.nz;
    hipMalloc(&h.sigma, vox * sizeof(float));
    hipMemcpy(h.sigma, desc.host_sigma, vox * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&h.rgb, vox * 3 * sizeof(float));
    hipMemcpy(h.rgb, desc.host_rgb, vox * 3 * sizeof(float), hipMemcpyHostToDevice);
    h.g_sigma = nullptr;
    h.g_rgb = nullptr;
    GridDenseDev* d = nullptr;
    hipMalloc(&d, sizeof(GridDenseDev));
    hipMemcpy(d, &h, sizeof(GridDenseDev), hipMemcpyHostToDevice);
    out.type = Field_Grid_Dense;
    out.device_ctx = d;
    return true;
}

void dvren::field_grid_dense_destroy(FieldProvider& fp)
{
    if (fp.type != Field_Grid_Dense || fp.device_ctx == nullptr) return;
    GridDenseDev h{};
    hipMemcpy(&h, fp.device_ctx, sizeof(GridDenseDev), hipMemcpyDeviceToHost);
    if (h.sigma) hipFree(h.sigma);
    if (h.rgb) hipFree(h.rgb);
    if (h.g_sigma) hipFree(h.g_sigma);
    if (h.g_rgb) hipFree(h.g_rgb);
    hipFree(fp.device_ctx);
    fp.device_ctx = nullptr;
}

void dvren::field_grid_dense_zero_grad(FieldProvider& fp)
{
    if (fp.type != Field_Grid_Dense || fp.device_ctx == nullptr) return;
    GridDenseDev h{};
    hipMemcpy(&h, fp.device_ctx, sizeof(GridDenseDev), hipMemcpyDeviceToHost);
    size_t vox = voxels_count(h);
    if (!h.g_sigma) hipMalloc(&h.g_sigma, vox * sizeof(float));
    if (!h.g_rgb) hipMalloc(&h.g_rgb, vox * 3 * sizeof(float));
    hipMemset(h.g_sigma, 0, vox * sizeof(float));
    hipMemset(h.g_rgb, 0, vox * 3 * sizeof(float));
    hipMemcpy(fp.device_ctx, &h, sizeof(GridDenseDev), hipMemcpyHostToDevice);
}

bool dvren::field_grid_dense_download_grad(const FieldProvider& fp, std::vector<float>& sigma_g, std::vector<float>& rgb_g)
{
    if (fp.type != Field_Grid_Dense || fp.device_ctx == nullptr) return false;
    GridDenseDev h{};
    hipMemcpy(&h, fp.device_ctx, sizeof(GridDenseDev), hipMemcpyDeviceToHost);
    if (!h.g_sigma || !h.g_rgb) return false;
    size_t vox = voxels_count(h);
    sigma_g.resize(vox);
    rgb_g.resize(vox * 3);
    hipMemcpy(sigma_g.data(), h.g_sigma, vox * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(rgb_g.data(), h.g_rgb, vox * 3 * sizeof(float), hipMemcpyDeviceToHost);
    return true;
}
